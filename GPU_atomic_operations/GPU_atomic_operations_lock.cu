// Warp counter in CUDA

#include <stdio.h>
#include <hip/hip_runtime.h>


// includes, project
////////////////////////////////////////////////////////////////////////////////
// declarations, forward

//extern "C"

// FILL HERE: define lock class
//            USe atomic operation for both lock and unlock functions
//            Fill free to use any atomic operation that correctly works for unlock.


/**
 * CUDA Kernel Device code
 * Computes cooperative additions
 */
// FILL HERE: Implement a kernel code that counts the total number of warps 
//            used in the kernel by using lock.


/**
 * Host main routine
 */
int 
main(void) 
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    int nwarps_host, *nwarps_dev;

    err = hipMalloc((void**)&nwarps_dev, sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device nwarps (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    nwarps_host = 0;

    err = hipMemcpy(nwarps_dev, &nwarps_host, sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy nwarps from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Kernel Invocation 
    int blocksPerGrid = 125;
    int threadsPerBlock = 1000;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    // FILL HERE: Defind a kernel invocation code that uses the blocksPerGrid blocks of threadsPerBlock threads


    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch blockCounterUnLocked kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();
	
    // Copy the device result to the host 
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(&nwarps_host, nwarps_dev, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy A from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();

    printf("number of warps = %d\n", nwarps_host);
    
    // Free device global memory
    hipFree(nwarps_dev);

    return 0;
}

